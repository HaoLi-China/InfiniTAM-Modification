#include "hip/hip_runtime.h"
#include "CUDA_KDtree.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <cstdio>

#define CUDA_STACK 100 // fixed size stack elements for each thread, increase as required. Used in SearchAtNodeRange.

void CheckCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


__device__ float Distance(const KDPoint &a, const KDPoint &b)
{
    float dist = 0;

    for(int i=0; i < KDTREE_DIM; i++) {
        float d = a.coords[i] - b.coords[i];
        dist += d*d;
    }

    return dist;
}

__device__ void SearchAtNode(const CUDA_KDNode *nodes, const int *indexes, const KDPoint *pts, int cur, const KDPoint &query, int *ret_index, float *ret_dist, int *ret_node)
{
    // Finds the first potential candidate

    int best_idx = 0;
    float best_dist = FLT_MAX;

    while(true) {
        int split_axis = nodes[cur].level % KDTREE_DIM;

        if(nodes[cur].left == -1) {
            *ret_node = cur;

            for(int i=0; i < nodes[cur].num_indexes; i++) {
                int idx = indexes[nodes[cur].indexes + i];
                float dist = Distance(query, pts[idx]);
                if(dist < best_dist) {
                    best_dist = dist;
                    best_idx = idx;
                }
            }

            break;
        }
        else if(query.coords[split_axis] < nodes[cur].split_value) {
            cur = nodes[cur].left;
        }
        else {
            cur = nodes[cur].right;
        }
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}


__device__ void SearchAtNodeRange(const CUDA_KDNode *nodes, const int *indexes, const KDPoint *pts, const KDPoint &query,int cur, float range, int *ret_index, float *ret_dist)
{
    // Goes through all the nodes that are within "range"

    int best_idx = 0;
    float best_dist = FLT_MAX;

    // Ok, we don't have nice STL vectors to use, and we can't dynamically allocate memory with CUDA??
    // We'll use a fixed length stack, increase this as required
    int to_visit[CUDA_STACK];
    int to_visit_pos = 0;

    to_visit[to_visit_pos++] = cur;

    while(to_visit_pos) {
        int next_search[CUDA_STACK];
        int next_search_pos = 0;

        while(to_visit_pos) {
            cur = to_visit[to_visit_pos-1];
            to_visit_pos--;

            int split_axis = nodes[cur].level % KDTREE_DIM;

            if(nodes[cur].left == -1) {
                for(int i=0; i < nodes[cur].num_indexes; i++) {
                    int idx = indexes[nodes[cur].indexes + i];
                    float d = Distance(query, pts[idx]);

                    if(d < best_dist) {
                        best_dist = d;
                        best_idx = idx;
                    }
                }
            }
            else {
                float d = query.coords[split_axis] - nodes[cur].split_value;

                // There are 3 possible scenarios
                // The hypercircle only intersects the left region
                // The hypercircle only intersects the right region
                // The hypercricle intersects both

                if(fabs(d) > range) {
                    if(d < 0)
                        next_search[next_search_pos++] = nodes[cur].left;
                    else
                        next_search[next_search_pos++] = nodes[cur].right;
                }
                else {
                    next_search[next_search_pos++] = nodes[cur].left;
                    next_search[next_search_pos++] = nodes[cur].right;
                }
            }
        }

        // No memcpy available??
        for(int i=0; i  < next_search_pos; i++)
            to_visit[i] = next_search[i];

        to_visit_pos = next_search_pos;
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}


__device__ void Search(const CUDA_KDNode *nodes, const int *indexes, const KDPoint *pts, const KDPoint &query, int *ret_index, float *ret_dist)
{
    // Find the first closest node, this will be the upper bound for the next searches
    int best_node = 0;
    int best_idx = 0;
    float best_dist = FLT_MAX;
    float radius = 0;

    SearchAtNode(nodes, indexes, pts, 0 /* root */, query, &best_idx, &best_dist, &best_node);

    radius = sqrt(best_dist);

    // Now find other possible candidates
    int cur = best_node;

    while(nodes[cur].parent != -1) {
        // Go up
        int parent = nodes[cur].parent;
        int split_axis = nodes[parent].level % KDTREE_DIM;

        // Search the other node
        float tmp_dist = FLT_MAX;
        int tmp_idx;

        if(fabs(nodes[parent].split_value - query.coords[split_axis]) <= radius) {
            // Search opposite node
            if(nodes[parent].left != cur)
                SearchAtNodeRange(nodes, indexes, pts, query, nodes[parent].left, radius, &tmp_idx, &tmp_dist);
            else
                SearchAtNodeRange(nodes, indexes, pts, query, nodes[parent].right, radius, &tmp_idx, &tmp_dist);
        }

        if(tmp_dist < best_dist) {
            best_dist = tmp_dist;
            best_idx = tmp_idx;
        }

        cur = parent;
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}

__global__ void SearchBatch(const CUDA_KDNode *nodes, const int *indexes, const KDPoint *pts, int num_pts, KDPoint *queries, int num_queries, int *ret_index, float *ret_dist)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx >= num_queries)
        return;

    Search(nodes, indexes, pts, queries[idx], &ret_index[idx], &ret_dist[idx]);
}

CUDA_KDTree::~CUDA_KDTree()
{
    hipFree(m_gpu_nodes);
    hipFree(m_gpu_indexes);
    hipFree(m_gpu_points);
}

void CUDA_KDTree::CreateKDTree(KDNode *root, int num_nodes, const vector <KDPoint> &data)
{
    // Create the nodes again on the CPU, laid out nicely for the GPU transfer
    // Not exactly memory efficient, since we're creating the entire tree again
    m_num_points = data.size();

    hipMalloc((void**)&m_gpu_nodes, sizeof(CUDA_KDNode)*num_nodes);
    hipMalloc((void**)&m_gpu_indexes, sizeof(int)*m_num_points);
    hipMalloc((void**)&m_gpu_points, sizeof(KDPoint)*m_num_points);

    CheckCUDAError("CreateKDTree");

    vector <CUDA_KDNode> cpu_nodes(num_nodes);
    vector <int> indexes(m_num_points);
    vector <KDNode*> to_visit;

    int cur_pos = 0;

    to_visit.push_back(root);

    while(to_visit.size()) {
        vector <KDNode*> next_search;

        while(to_visit.size()) {
            KDNode *cur = to_visit.back();
            to_visit.pop_back();

            int id = cur->id;

            cpu_nodes[id].level = cur->level;
            cpu_nodes[id].parent = cur->_parent;
            cpu_nodes[id].left = cur->_left;
            cpu_nodes[id].right = cur->_right;
            cpu_nodes[id].split_value = cur->split_value;
            cpu_nodes[id].num_indexes = cur->indexes.size();

            if(cur->indexes.size()) {
                for(unsigned int i=0; i < cur->indexes.size(); i++)
                    indexes[cur_pos+i] = cur->indexes[i];

                cpu_nodes[id].indexes = cur_pos;
                cur_pos += cur->indexes.size();
            }
            else {
                cpu_nodes[id].indexes = -1;
            }

            if(cur->left)
                next_search.push_back(cur->left);

            if(cur->right)
                next_search.push_back(cur->right);
        }

        to_visit = next_search;
    }

    hipMemcpy(m_gpu_nodes, &cpu_nodes[0], sizeof(CUDA_KDNode)*cpu_nodes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_indexes, &indexes[0], sizeof(int)*indexes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_points, &data[0], sizeof(KDPoint)*data.size(), hipMemcpyHostToDevice);

    CheckCUDAError("CreateKDTree");
}

void CUDA_KDTree::Search(const vector <KDPoint> &queries, vector <int> &indexes, vector <float> &dists)
{
    int threads = 512;
    int blocks = queries.size()/threads + ((queries.size() % threads)?1:0);

    KDPoint *gpu_queries;
    int *gpu_ret_indexes;
    float *gpu_ret_dist;

    indexes.resize(queries.size());
    dists.resize(queries.size());

    hipMalloc((void**)&gpu_queries, sizeof(KDPoint)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_indexes, sizeof(int)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_dist, sizeof(float)*queries.size()*KDTREE_DIM);

    CheckCUDAError("Search");

    hipMemcpy(gpu_queries, &queries[0], sizeof(float)*queries.size()*KDTREE_DIM, hipMemcpyHostToDevice);

    CheckCUDAError("Search");

    printf("CUDA blocks/threads: %d %d\n", blocks, threads);

    SearchBatch<<<blocks, threads>>>(m_gpu_nodes, m_gpu_indexes, m_gpu_points, m_num_points, gpu_queries, queries.size(), gpu_ret_indexes, gpu_ret_dist);
    hipDeviceSynchronize();

    CheckCUDAError("Search");

    hipMemcpy(&indexes[0], gpu_ret_indexes, sizeof(int)*queries.size(), hipMemcpyDeviceToHost);
    hipMemcpy(&dists[0], gpu_ret_dist, sizeof(float)*queries.size(), hipMemcpyDeviceToHost);

    hipFree(gpu_queries);
    hipFree(gpu_ret_indexes);
    hipFree(gpu_ret_dist);
}
